#include "hip/hip_runtime.h"
//Seung Lee - A01021720
//Matrix Mult con tiles
//Reutilizamos un poco del codigo del a-1

#include "common.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <chrono>
#include <iostream>

using namespace std;

//Profe leer el reporte para saber porque lo deje en 1700 :(
#define NSize 1700 //Definimos el tamano de nuestra matriz N x N
#define TILESize 32 //Cambiamos aqui el TILESize para que se cambie para todo el codigo

void fillMat(float * ip, const int size) { //Funcion para llenar nuestras matrices (hecho como el ejemplo en clase matrix_sum_1d)
    for(int i = 0; i < size; i++) {
        ip[i] = (rand() / (float)RAND_MAX * 10.0f); //Llenarlos con puros random floats entre 0 y 10
        // ip[i] = i;
    }
}

// grid 1D block 1D
__global__ void multMatrixOnGPU2D(float *A, float *B, float *C, int nx, int ny)
{
    //Codigo de clase
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        for(int i = 0; i < ny; i++) {
            C[ix*ny+iy] += A[ix*ny+i] * B[i*ny+iy];
        }
    }
    // float temp = 0;
    // if (ix < nx && iy < ny){
    //     for(int i = 0; i < nx; i++) {
    //         for(int j = 0; j < ny; j++) {
    //             temp += (A[ix * nx + i] * B[ny + iy * j]);
    //         }
    //     }
    //     C[idx] = temp;
    // }
}

//Funcion de matrix mult con tiles
__global__ void multMatrixOnTiles(float *A, float *B, float *C, int nx, int ny) {
    //Codigo de clase
    unsigned int ix = threadIdx.x + blockIdx.x * TILESize;
    unsigned int iy = threadIdx.y + blockIdx.y * TILESize;

    //Codigo de class demos
    // __shared__ int s[256];
    // int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float matTempA[TILESize][TILESize];
    __shared__ float matTempB[TILESize][TILESize];

    //Llenamos las matrices shared y las inicializamos con puros 0's
    for(int i = 0; i < TILESize; i ++) {
        for(int j = 0; j < TILESize; j++) {
            matTempA[i][j] = 0;
            matTempB[i][j] = 0;
        }
    }

    float temp = 0;

    //vamos a traves de todos los tiles
    for(int i = (TILESize + nx - 1)/TILESize; i >= 0; i--) {
        if((i * TILESize + threadIdx.x) < nx && (iy < ny)) {
            matTempA[threadIdx.y][threadIdx.x] = A[(iy*ny) + (i*TILESize+threadIdx.x)];
        }

        if((i * TILESize + threadIdx.y) < ny && (ix < nx)) {
            matTempB[threadIdx.y][threadIdx.x] = B[(i*TILESize+threadIdx.y) * nx + ix];
        }

        __syncthreads(); //Tenemos que utilizar syncthreads despues de modificar las matrices en threadIdx

        for(int j = 0; j < TILESize; j++) {
            temp += matTempA[threadIdx.y][j] * matTempB[j][threadIdx.x];
        }

        __syncthreads();
    }

    // if (ix < nx && iy < ny) {
    //     for(int i = 0; i < ny; i++) {
    //         C[ix*ny+iy] += A[ix*ny+i] * B[i*ny+iy];
    //     }
    // }
    if(ix < nx && iy < ny) {
        C[ix*ny+iy] = temp;
    }
}

void multMat(float *A, float *B, float *C, const int nx, const int ny) { //Funcion para multiplicar matriz (como ejemplo)
    for(int i = 0; i < ny; i++) {
        for(int j = 0; j < nx; j++) {
            for(int k = 0; k < ny; k++) { //Regla del karatazo pu pi pao
                C[i * nx + j] += (A[i * nx + k] * B[k + nx * j]);
                // printf("G"); //Debug
            }
        }
    }
}

//Checar resultado
void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N*N; i++)
    {
        if (fabs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Matrix multiplications from host and GPU match!.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // set up data size of matrix
    int nx = NSize;
    int ny = NSize;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // Inicializar nuestros datos
    fillMat(h_A, nxy);
    fillMat(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result SAFE_CALLs
    auto start_cpu =  chrono::high_resolution_clock::now();
    multMat(h_A, h_B, hostRef, nx, ny);
    auto end_cpu =  chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("MultMat en Host elapsed %f ms\n\n", duration_ms.count());

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = TILESize;
    int dimy = TILESize;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    //MULTMAT ON GPU 2D_2D (ya se tenia)
    //Multiplicar matrices con cantidad de repeticiones
    int timeAverage = 0;
    // add matrix at host side for result SAFE_CALLs
    //Lo sacamos del ejemplo de clase
    start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  chrono::high_resolution_clock::now();
    duration_ms = end_cpu - start_cpu;
    timeAverage += duration_ms.count();
    int performanceTime = timeAverage;
    printf("La cantidad de tiempo que se tarda cada ejecucion con GPU con threads es alrededor de: %d ms\n", performanceTime);
    printf("Tamano de matriz: %d x %d\n", nx, ny);

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    //MULTMAT CON TILING GPU
    timeAverage = 0;
    // add matrix at host side for result SAFE_CALLs
    //Lo sacamos del ejemplo de clase
    start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnTiles<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  chrono::high_resolution_clock::now();
    duration_ms = end_cpu - start_cpu;
    timeAverage += duration_ms.count();
    performanceTime = timeAverage;
    printf("La cantidad de tiempo que se tarda cada ejecucion con TILING de %d x %d es alrededor de: %d ms\n", TILESize, TILESize, performanceTime);
    printf("Tamano de matriz: %d x %d\n", nx, ny);

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return (0);
}